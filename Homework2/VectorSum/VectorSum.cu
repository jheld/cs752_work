
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

__global__ void vectorSum(float *a, float *b, float *c){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  c[i] = a[i] + b[i];
}

int main(int argc, char *argv[]){
  unsigned int length = 4194304;
  int i, Size;
  float *a, *b, *c, *copyC, *gpuA, *gpuB, *gpuC;
  time_t seed;
  hipEvent_t start;
  hipEvent_t stop;
  float msecTotal;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  if (argc>1) {
    sscanf(argv[1],"%d",&length);
  }
  Size = sizeof(float)*length;
  a = (float *)malloc(Size);
  b = (float *)malloc(Size);
  c = (float *)malloc(Size);
  copyC = (float *)malloc(Size);
  time(&seed);
  srand48(seed);
  for (i=0; i<length; i++)
    a[i] = drand48(), b[i] = drand48();

  hipSetDevice(0);
  hipMalloc((void**)&gpuA, Size);
  hipMalloc((void**)&gpuB, Size);
  hipMalloc((void**)&gpuC, Size);

  hipEventRecord(start, NULL);
  for (i=0; i<length; i++)
    c[i] = a[i] + b[i];
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msecTotal, start, stop);
  printf("cpu time: %.3f ms\n", msecTotal);

  hipMemcpy(gpuA, a, Size, hipMemcpyHostToDevice);
  hipMemcpy(gpuB, b, Size, hipMemcpyHostToDevice);
  dim3 numThreads(512, 1);
  dim3 numBlocks(32, 1);
  hipEventRecord(start, NULL);
  vectorSum<<<numBlocks, numThreads>>>(gpuA, gpuB, gpuC);
  hipDeviceSynchronize();
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipMemcpy(copyC, gpuC, Size, hipMemcpyDeviceToHost);

  hipEventElapsedTime(&msecTotal, start, stop);
  printf("gpu time: %.3f ms\n", msecTotal);

  for (i=0; i<length; i++)
    if (fabs(c[i]-copyC[i]) > 0.000001){
      printf("%d\t%f\t%f\n", i, c[i], copyC[i]);
      return 1;
    }
  return 0;
}
