
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

__global__ void vectorSum(float *a, float *b, float *c){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  c[i] = a[i] + b[i];
}

int main(int argc, char *argv[]){
  unsigned int length = 4194304;
  int i, Size;
  float *a, *b, *c, *copyC, *gpuA, *gpuB, *gpuC;
  time_t seed;
  hipEvent_t start;
  hipEvent_t stop;
  float msecTotal;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  if (argc>1)
    sscanf(argv[1],"%d",&length);
  Size = sizeof(float)*length;
  unsigned long int padded_length = floor((length + ((512*32)-1))/(1.0*512*32)) * (1.0*512*32);
  a = (float *)calloc(length, sizeof(float));
  b = (float *)calloc(length, sizeof(float));
  c = (float *)calloc(length, sizeof(float));
  copyC = (float *)calloc(length, sizeof(float));
  time(&seed);
  srand48(seed);
  for (i=0; i<length; i++)
    a[i] = drand48(), b[i] = drand48();
  hipSetDevice(0);
  hipError_t error;
  error = hipMalloc((void**)&gpuA, padded_length*sizeof(float));
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }
  error = hipMemset(gpuA, 0, padded_length*sizeof(float));
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }
  error = hipMalloc((void**)&gpuB, padded_length*sizeof(float));
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }
  error = hipMemset(gpuB, 0, padded_length*sizeof(float));
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }
  error = hipMalloc((void**)&gpuC, padded_length*sizeof(float));
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }
  error = hipMemset(gpuC, 0, padded_length*sizeof(float));
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }

  hipEventRecord(start, NULL);
  for (i=0; i<length; i++)
    c[i] = a[i] + b[i];
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msecTotal, start, stop);
  printf("cpu time: %.3f ms\n", msecTotal);
  error = hipMemcpy(gpuA, a, Size, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }

  error = hipMemcpy(gpuB, b, Size, hipMemcpyHostToDevice);
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }

  dim3 numThreads(512, 1);
  dim3 numBlocks(32, 1);
  hipEventRecord(start, NULL);
  vectorSum<<<numBlocks, numThreads>>>(gpuA, gpuB, gpuC);
  hipDeviceSynchronize();
  hipEventRecord(stop, NULL);
  hipEventSynchronize(stop);
  error = hipMemcpy(copyC, gpuC, Size, hipMemcpyDeviceToHost);
  if (error != hipSuccess) {
    printf("oops, %d, error: %d\n", __LINE__, error);
    exit(EXIT_FAILURE);
  }

  hipEventElapsedTime(&msecTotal, start, stop);
  printf("gpu time: %.3f ms\n", msecTotal);

  for (i=0; i<length; i++)
    if (fabs(c[i]-copyC[i]) > 0.000001){
      printf("%d\t%f\t%f\n", i, c[i], copyC[i]);
      return 1;
    }
  return 0;
}
